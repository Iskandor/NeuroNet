#include "hip/hip_runtime.h"
#include "NeuralGroup.cuh"
#include "network_kernel.cuh"
#include "vector_kernel.cuh"
#include "Define.h"

#include <memory>

using namespace std;

NeuralGroup::NeuralGroup(int p_id, int p_dim, int p_activationFunction)
{
  _id = p_id;
  _dim = p_dim;
  _activationFunction = p_activationFunction;

  _output = new double[p_dim];
  memset(_output, 0, sizeof(double)*p_dim);

  if (_activationFunction == BIAS) {
    for(int i = 0; i < p_dim; i++) {
      _output[i] = 1;
    }
  }

  _actionPotential = new double[p_dim];
  memset(_actionPotential, 0, sizeof(double)*p_dim);

  _valid = false;
}


NeuralGroup::~NeuralGroup(void)
{
	delete[] _actionPotential;
  delete[] _output;
}

void NeuralGroup::init() {
}

/* calculate output of group */
void NeuralGroup::fire() {
    _valid = true;
    activate(_actionPotential, _activationFunction);
}

void NeuralGroup::addInConnection(int p_index) {
    _inConnections.push_back(p_index);
}

void NeuralGroup::addOutConnection(int p_index) {
    _outConnections.push_back(p_index);
}

/* wrapper for CUDA function which will calculate the sum of inputs for each neuron */
hipError_t NeuralGroup::integrate(double *p_input, double *p_weights, int p_input_dim) {
  hipError_t cudaStatus;
  
  double *dev_input = 0;
  double *dev_weights = 0;
  double *dev_output = 0;
  double *dev_ac = 0;

  cudaStatus = hipMalloc((void**)&dev_input, p_input_dim * sizeof(double));
  cudaStatus = hipMalloc((void**)&dev_weights, _dim * p_input_dim * sizeof(double));
  cudaStatus = hipMalloc((void**)&dev_output, _dim * sizeof(double));
  cudaStatus = hipMalloc((void**)&dev_ac, _dim * sizeof(double));
  cudaStatus = hipMemcpy(dev_ac, _actionPotential, _dim * sizeof(double), hipMemcpyHostToDevice);

  cudaStatus = hipMemcpy(dev_input, p_input, p_input_dim * sizeof(double), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpy(dev_weights, p_weights, _dim * p_input_dim * sizeof(double), hipMemcpyHostToDevice);

  integrateKernel<<<(int)ceil((double)_dim/(double)MAX_THREAD),_dim>>>(dev_output, dev_input, dev_weights, p_input_dim);
  cudaStatus = hipGetLastError();
  cudaStatus = hipDeviceSynchronize();
    
  addVectorKernel<<<(int)ceil((double)_dim/(double)MAX_THREAD),_dim>>>(dev_ac, dev_output);
  cudaStatus = hipGetLastError();
  cudaStatus = hipDeviceSynchronize();

  cudaStatus = hipMemcpy(_actionPotential, dev_ac, _dim * sizeof(double), hipMemcpyDeviceToHost);

  cudaStatus = hipFree(dev_input);
  cudaStatus = hipFree(dev_weights);
  cudaStatus = hipFree(dev_output);
  cudaStatus = hipFree(dev_ac);

	return cudaStatus;
}

/* function which should calculate the output of neuron (activation function output) according to action potential */
hipError_t NeuralGroup::activate(double* p_input, const int p_activationFunction) {
  hipError_t cudaStatus;

  int    *dev_activationFunction = 0;
  double *dev_input = 0;
  double *dev_output = 0;
  cudaStatus = hipMalloc((void**)&dev_activationFunction, sizeof(int));
  cudaStatus = hipMalloc((void**)&dev_input, _dim * sizeof(double));
  cudaStatus = hipMalloc((void**)&dev_output, _dim * sizeof(double));

  cudaStatus = hipMemcpy(dev_input, p_input, _dim * sizeof(double), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpy(dev_activationFunction, &p_activationFunction, sizeof(int), hipMemcpyHostToDevice);
  activateKernel<<<(int)ceil((double)_dim/(double)MAX_THREAD),_dim>>>(dev_output, dev_input, dev_activationFunction);
  cudaStatus = hipGetLastError();
  cudaStatus = hipDeviceSynchronize();
  cudaStatus = hipMemcpy(_output, dev_output, _dim * sizeof(double), hipMemcpyDeviceToHost);
  cudaStatus = hipMemcpy(p_input, dev_input, _dim * sizeof(double), hipMemcpyDeviceToHost);

  cudaStatus = hipFree(dev_activationFunction);
  cudaStatus = hipFree(dev_input);
  cudaStatus = hipFree(dev_output);

  return cudaStatus;
}